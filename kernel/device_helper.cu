#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <device_types.h>
#include <hip/driver_types.h>
#include <cstdio>
#include <iostream>

#include "color.hpp"
#include "device_helper.hpp"

__global__ void add(int n, float* x, float* y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) y[i] = x[i] + y[i];
}

__global__ void init_rand_states(cu_camera* d_cam) { d_cam->initRandState(); }

__global__ void device_render(cu_hittable** d_world, cu_camera* d_cam,
                              color3* d_ouput) {
    printf("Block: %d, Thread: %d", blockIdx.x, threadIdx.x);
    if (threadIdx.x !=0 || blockIdx.x != 0) return;
    int width = d_cam->image_width;
    int height = d_cam->image_height;

    printf("World ptr ptr: %p\n", d_world);
    printf("World ptr: %p\n", *d_world);
    printf("%d X %d\n", width, height);

    auto r = d_cam->get_ray(0, 0);
    auto h = cu_hit_record();
    (*d_world)->hit(r, interval(0.001, inf), h);

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            auto ray = d_cam->get_ray(i, j);
            printf("Ray: %f, %f, %f\n", ray.direction().x(), ray.direction().y(),
                   ray.direction().z());
            printf("%d, %d\n", i, j);
            auto op = d_cam->ray_color(ray, *d_world, d_cam->max_depth);
            d_ouput[i * width + j] = op;
        }
    }
}

void render(cu_camera& cam, color3* output, cu_hittable** world) {
    std::clog << "Hello Render" << std::endl;
    cu_camera* d_cam;

    hipError_t err = hipMallocManaged(&d_cam, sizeof(cu_camera));
    if (err != hipSuccess) {
        std::clog << "Could not allocate camera on the GPU" << std::endl;
        return;
    }

    *d_cam = cam;

    init_rand_states<<<1, 1>>>(d_cam);
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::clog << "Could not initialize random state" << std::endl;
        return;
    }

    color3* d_output;
    err = hipMallocManaged(
        &d_output, cam.image_width * cam.image_height * sizeof(color3));
    if (err != hipSuccess) {
        std::clog << "Could not allocate camera on the GPU" << std::endl;
    }

    device_render<<<1, 1>>>(world, d_cam, d_output);
    hipDeviceSynchronize();

    for (int i = 0; i < cam.image_height * cam.image_width; i++) {
        // std::clog<<d_output[i]<<std::endl;
        output[i] = d_output[i];
    }

    hipFree(d_output);
    hipFree(d_cam);
}

void run() {
    int N = 1 << 20;
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
}
