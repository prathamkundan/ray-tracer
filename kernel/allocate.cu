#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include <iostream>
#include "color.hpp"
#include "cuda/cu_camera.hpp"
#include "cuda/cu_hittable.hpp"
#include "cuda/cu_hittable_list.hpp"
#include "cuda/cu_material.hpp"
#include "cuda/cu_sphere.hpp"
#include "cuda/cu_allocate.hpp"
#include "utils.hpp"

__global__ void cu_allocate_sphere(const point3* center, double radius,
                                   cu_material** mat,
                                   cu_hittable** sphere_ptr) {
    *sphere_ptr = new cu_sphere(*center, radius, *mat);
}

cu_hittable** Allocator::allocate_sphere(const point3 center, double radius,
                                         cu_material** mat) {
    cu_hittable** d_sphere;

    auto err = hipMallocManaged(&d_sphere, sizeof(cu_hittable*));
    if (err != hipSuccess) {
        std::cerr << "Could not allocate sphere::hipMalloc failed"
                  << std::endl;
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }

    vec3* d_center;
    err = hipMallocManaged(&d_center, sizeof(vec3));
    if (err != hipSuccess) {
        std::cerr << "Could not allocate vec3::hipMalloc failed" << std::endl;
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }
    *d_center = center;

    cu_allocate_sphere<<<1, 1>>>(d_center, radius, mat, d_sphere);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Could not construct sphere on device" << std::endl;
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }

    allocated_hittables.push_back(d_sphere);
    return d_sphere;
}

__global__ void cu_allocate_list(cu_hittable*** d_objects, int d_num_objects,
                                 cu_hittable** hittable_list_ptr) {
    cu_hittable_list* new_ptr = new cu_hittable_list();
    new_ptr->set_objects(d_objects, d_num_objects);
    *hittable_list_ptr = new_ptr;
}

__global__ void render(cu_hittable** d_world, cu_camera* d_cam,
                       color3* d_output) {
    d_cam->initRandState();

    int width = d_cam->image_width;
    int height = d_cam->image_height;

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            ray ray = d_cam->get_ray(i, j);
            // printf("Ray: %f, %f, %f\n", ray.direction().x(),
            // ray.direction().y(),
            //        ray.direction().z());
            printf("%d, %d\n", i, j);
            color3 op = d_cam->ray_color_iter(ray, *d_world, d_cam->max_depth);
            d_output[i * width + j] = op;
        }
    }
}

__global__ void render_parallel(cu_hittable** d_world, cu_camera* d_cam,
                                color3* d_output) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int width = d_cam->image_width;
    int height = d_cam->image_height;

    // printf("DIM: %d, %d\n", x, y);

    if (x >= height || y >= width) return;

    color3 op = color3(0, 0, 0);
    for (int i = 0; i < d_cam->samples_per_pixel; i++) {
        auto ray = d_cam->get_ray(y, x);
        op += d_cam->ray_color_iter(ray, *d_world, d_cam->max_depth);
    }

    op *= d_cam->pixel_samples_scale;
    d_output[x * width + y] = op;
}

void Allocator::test(cu_camera cam, color3* output) {
    std::clog << "Hello Render" << std::endl;
    cu_camera* d_cam;

    hipError_t err = hipMallocManaged(&d_cam, sizeof(cu_camera));
    if (err != hipSuccess) {
        std::clog << "Could not allocate camera on the GPU" << std::endl;
        return;
    }

    *d_cam = cam;

    color3* d_output;
    err = hipMallocManaged(
        &d_output, cam.image_width * cam.image_height * sizeof(color3));
    if (err != hipSuccess) {
        std::clog << "Could not allocate camera on the GPU" << std::endl;
    }

    dim3 threads_per_block(16, 16);  // 16x16 threads per block
    dim3 number_of_blocks(ceil(cam.image_height / 16.0),
                          ceil(cam.image_width / 16.0));

    render_parallel<<<number_of_blocks, threads_per_block>>>(world, d_cam,
                                                             d_output);
    // render<<<1, 1>>>(world, d_cam, d_output);
    err = hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    for (int i = 0; i < cam.image_height * cam.image_width; i++) {
        // std::clog<<d_output[i]<<std::endl;
        output[i] = d_output[i];
    }

    hipFree(d_cam);
    hipFree(d_output);
}

cu_hittable** Allocator::allocate_list() {
    cu_hittable** d_hittable_list;
    auto err = hipMallocManaged(&d_hittable_list, sizeof(cu_hittable*));
    if (err != hipSuccess) {
        std::cerr << "Could not allocate hittable_list ::hipMalloc failed"
                  << std::endl;
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }

    cu_hittable*** d_objects;
    hipMallocManaged(&d_objects,
                      allocated_hittables.size() * sizeof(cu_hittable**));

    for (int i = 0; i < allocated_hittables.size(); i++) {
        d_objects[i] = allocated_hittables[i];
    }

    cu_allocate_list<<<1, 1>>>(d_objects, allocated_hittables.size(),
                               d_hittable_list);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Could not construct hittable_list" << std::endl;
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }

    this->world = d_hittable_list;
    return d_hittable_list;
}

__global__ void cu_allocate_lambertian(color3* albedo,
                                       cu_material** material_ptr) {
    *material_ptr = new cu_lambertian(*albedo);
}

cu_material** Allocator::allocate_lambertian(color3 albedo) {
    cu_material** lambertian_ptr;

    auto err = hipMallocManaged(&lambertian_ptr, sizeof(cu_material*));
    if (err != hipSuccess) {
        std::clog
            << "Could not allocate material:lambertian ::hipMalloc failed"
            << std::endl;
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }

    color3* d_color;
    hipMallocManaged(&d_color, sizeof(color3));
    *d_color = albedo;

    cu_allocate_lambertian<<<1, 1>>>(d_color, lambertian_ptr);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Could not construct material:lambertian" << std::endl;
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }

    allocated_materials.push_back(lambertian_ptr);
    return lambertian_ptr;
}

__global__ void cu_allocate_metal(color3* albedo, double fuzz,
                                  cu_material** material_ptr) {
    *material_ptr = new cu_metal(*albedo, fuzz);
}

cu_material** Allocator::allocate_metal(const color3 albedo, double fuzz) {
    cu_material** metal_ptr;

    auto err = hipMallocManaged(&metal_ptr, sizeof(cu_material*));
    if (err != hipSuccess) {
        std::cerr << "Could not allocate item::hipMalloc failed" << std::endl;
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }

    color3* d_color;
    hipMallocManaged(&d_color, sizeof(color3));
    *d_color = albedo;

    cu_allocate_metal<<<1, 1>>>(d_color, fuzz, metal_ptr);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Could not allocate item" << std::endl;
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }

    allocated_materials.push_back(metal_ptr);
    return metal_ptr;
}

__global__ void cu_allocate_dielectric(double refraction_index,
                                       cu_material** material_ptr) {
    *material_ptr = new cu_dielectric(refraction_index);
}

cu_material** Allocator::allocate_dielectric(double refraction_index) {
    cu_material** dielectric_ptr;

    auto err = hipMallocManaged(&dielectric_ptr, sizeof(cu_material*));
    if (err != hipSuccess) {
        std::cerr << "Could not allocate item::hipMalloc failed" << std::endl;
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }

    cu_allocate_dielectric<<<1, 1>>>(refraction_index, dielectric_ptr);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Could not allocate item" << std::endl;
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }

    allocated_materials.push_back(dielectric_ptr);
    return dielectric_ptr;
}
